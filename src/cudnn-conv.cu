#include <hipDNN.h>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#include "imgutils.h"

#define checkCUDNN(expression)                             \
{                                                          \
    hipdnnStatus_t status = (expression);                   \
    if (status != HIPDNN_STATUS_SUCCESS) {                  \
        std::cerr << "Error on line " << __LINE__ << ": "  \
        << hipdnnGetErrorString(status) << std::endl;       \
        std::exit(EXIT_FAILURE);                           \
    }                                                      \
}

#define ITERATIONS 16
#define KERNEL_SIZE 5
#define KERNEL_RADIUS ((KERNEL_SIZE - 1) / 2)

int main(int argc, const char* argv[]) {
    if (argc < 2) {
        std::cerr << "usage: conv <image> [gpu=0]" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    int gpu_id = (argc > 2) ? std::atoi(argv[2]) : 0;
    std::cerr << "GPU: " << gpu_id << std::endl;

    cv::Mat image   = read_image_bw(argv[1]);
    int im_channels = image.channels();

    hipSetDevice(gpu_id);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                /*format=*/HIPDNN_TENSOR_NHWC,
                /*dataType=*/HIPDNN_DATA_FLOAT,
                /*batch_size=*/1,
                /*channels=*/im_channels,
                /*image_height=*/image.rows,
                /*image_width=*/image.cols));

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                /*dataType=*/HIPDNN_DATA_FLOAT,
                /*format=*/HIPDNN_TENSOR_NCHW,
                /*out_channels=*/im_channels,
                /*in_channels=*/im_channels,
                /*kernel_height=*/KERNEL_SIZE,
                /*kernel_width=*/KERNEL_SIZE));

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                /*pad_height=*/KERNEL_RADIUS,
                /*pad_width=*/KERNEL_RADIUS,
                /*vertical_stride=*/1,
                /*horizontal_stride=*/1,
                /*dilation_height=*/1,
                /*dilation_width=*/1,
                /*mode=*/HIPDNN_CROSS_CORRELATION,
                /*computeType=*/HIPDNN_DATA_FLOAT));

    int batch_size{0}, channels{0}, height{0}, width{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                input_descriptor,
                kernel_descriptor,
                &batch_size,
                &channels,
                &height,
                &width));

    std::cerr << "Output Image: " << height << " x " << width << " x " << channels
        << std::endl;

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                /*format=*/HIPDNN_TENSOR_NHWC,
                /*dataType=*/HIPDNN_DATA_FLOAT,
                /*batch_size=*/1,
                /*channels=*/channels,
                /*image_height=*/image.rows,
                /*image_width=*/image.cols));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(
            hipdnnGetConvolutionForwardAlgorithm(cudnn,
                input_descriptor,
                kernel_descriptor,
                convolution_descriptor,
                output_descriptor,
                HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                /*memoryLimitInBytes=*/0,
                &convolution_algorithm));

    size_t workspace_bytes{0};
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                input_descriptor,
                kernel_descriptor,
                convolution_descriptor,
                output_descriptor,
                convolution_algorithm,
                &workspace_bytes));
    std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
        << std::endl;
    assert(workspace_bytes > 0);

    void* d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    int image_bytes = batch_size * channels * height * width * sizeof(float);

    float* d_input{nullptr};
    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, image.ptr<float>(0), image_bytes, hipMemcpyHostToDevice);

    float* d_output{nullptr};
    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output, 0, image_bytes);

    // clang-format off
    //const float kernel_template[KERNEL_SIZE][KERNEL_SIZE] = {
    //    {1, 1, 1},
    //    {1, -8, 1},
    //    {1, 1, 1}
    //};
    // clang-format on

    float h_kernel[1][channels][KERNEL_SIZE][KERNEL_SIZE];
    for (int channel = 0; channel < channels; ++channel) {
        for (int row = 0; row < KERNEL_SIZE; ++row) {
            for (int column = 0; column < KERNEL_SIZE; ++column) {
                h_kernel[0][channel][row][column] = (float)(rand() % 16);
            }
        }
    }

    float* d_kernel{nullptr};
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

    const float alpha = 1.0f, beta = 0.0f;

    StopWatchInterface *hTimer = NULL;
    sdkCreateTimer(&hTimer);

    std::cerr << "Running convolutions..." << std::endl;

    for (int i = -1; i < ITERATIONS; ++i) {
        if (i == 0) {
            checkCudaErrors(hipDeviceSynchronize());
            sdkResetTimer(&hTimer);
            sdkStartTimer(&hTimer);
        }

        checkCUDNN(hipdnnConvolutionForward(cudnn,
                    &alpha,
                    input_descriptor,
                    d_input,
                    kernel_descriptor,
                    d_kernel,
                    convolution_descriptor,
                    convolution_algorithm,
                    d_workspace,
                    workspace_bytes,
                    &beta,
                    output_descriptor,
                    d_output));
    }

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    double time = sdkGetTimerValue(&hTimer) / (double)ITERATIONS;
    printf("Kernel time = %.5f ms\n", time);

    float* h_output = new float[image_bytes];
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);

    save_image_bw("cudnn-out.png", h_output, height, width);

    delete[] h_output;
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);
}
