#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <cmath>

#include "imgutils.h"

#define RADIUS 1
#define FILTER_SIZE ((RADIUS * 2) + 1)
#define ITERATIONS 128

#define PRINT 1
#define RANDOM 1

__constant__ float c_filter[FILTER_SIZE*FILTER_SIZE];

__global__ void kernel(float* d_in, int height, int width, float* d_out) {

    __shared__ float sh_out[blockDim.z];

    // Get global position in grid
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = threadIdx.z;

    // actual location within image data
    // since image data is interleaved RGB values, offset like you would a 2D
    // image, multiply that by the number of channels (3) and add the z value
    // representing whether the pixel is R, G, or B
    unsigned int loc = (y * width) + x;

    // only perform convolution on pixels within radius
    if (x >= RADIUS && y >= RADIUS && x < (width - RADIUS) && y < (height - RADIUS) && z < FILTER_SIZE*FILTER_SIZE) {
        atomicAdd(&d_out[loc], d_in[loc] * c_filter[z]);
    }
}

int main(int argc, char** argv) {
    if (argc < 2) {
        printf("Usage: ./naive-conv <image>\n");
        return 0;
    }

    // read in image
    cv::Mat h_in     = read_image_bw(argv[1]);
    int     height   = h_in.rows;
    int     width    = h_in.cols;

    // Declare image and filter variables for host and device
    float *h_filter, *h_out, *d_in, *d_out;

    // size to allocate for image and filter variables
    unsigned int img_size         = width * height * sizeof(float);
    unsigned int full_filter_size = FILTER_SIZE * FILTER_SIZE * sizeof(float);

#if PRINT
    printf("img_size=%u, full_filter_size=%u\n", img_size, full_filter_size);
#endif

    // Allocate host data
    h_filter = (float*)malloc(full_filter_size);
    h_out    = (float*)malloc(img_size);

    // copy filter template to actual filter (maybe redundant)
#if RANDOM
    srand(200);
#else
    // Initialize filter template
    // clang-format off
    const float filt_template[FILTER_SIZE][FILTER_SIZE] = {
        {1,   1, 1},
        {1,  -8, 1},
        {1,   1, 1}
    };
    // clang-format on
#endif

    for (int row = 0; row < FILTER_SIZE; ++row) {
        for (int col = 0; col < FILTER_SIZE; ++col) {
            int idx = row * FILTER_SIZE + col;
#if RANDOM
            h_filter[idx] = (float)(rand() % 16);
#else
            h_filter[idx] = filt_template[row][col];
#endif
        }
    }

    // Allocate device data
    checkCudaErrors(hipMalloc((void**)&d_in, img_size));
    checkCudaErrors(hipMalloc((void**)&d_out, img_size));

    // Copy host memory to device
    checkCudaErrors(hipMemcpy(d_in, h_in.data, img_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_filter), h_filter, full_filter_size));

    // calculate next power of 2 up from # elements in filter
    // this will help make the number of threads a power of 32
    double log2FS = log2((double)(FILTER_SIZE*FILTER_SIZE));
    double ceilFS = ceil(log2FS);
    int nextpow2 = (int)pow(2, ceilFS);

    // calculate block size
    int maxNumThreads = 1024; // max threads per block
    int maxFullBS = maxNumThreads/nextpow2; // max val of blockSize.x*blockSize.y
    double maxBS = sqrt(maxFullBS); // max possible blockSize
    double log2BS = log2(maxBS);          //
    double floorLog = floor(log2BS);      // prev power of 2 from max block size
    int prevpow2 = (int)pow(2, floorLog); //
    int block_size = prevpow2;
    //int block_size = (int)sqrt((1024.0 / (double)(FILTER_SIZE*FILTER_SIZE)));
    //int nextpow2 = FILTER_SIZE*FILTER_SIZE;
    
    // Let grid size be based on block size
    // Have just enough blocks to cover whole image
    // The -1 is to cover the case where image dimensions are multiples of
    // BLOCKS_SIZE
    int gridXSize = 1 + ((width - 1) / block_size);
    int gridYSize = 1 + ((height - 1) / block_size);
#if PRINT
    printf("gridXSize=%d, gridYSize=%d, nextpow2=%d, block_size=%d\n", 
            gridXSize, gridYSize, nextpow2, block_size);
#endif
    dim3 h_gridDim(gridXSize, gridYSize);
    dim3 h_blockDim(block_size, block_size, nextpow2);

    // Run on GPU 0
    hipSetDevice(0);

    // Timing stuff
    StopWatchInterface *hTimer = NULL;
    sdkCreateTimer(&hTimer);

    // Kernel call
    for (int i = -1; i < ITERATIONS; ++i) {

        if (i == 0) {
            checkCudaErrors(hipDeviceSynchronize());
            sdkResetTimer(&hTimer);
            sdkStartTimer(&hTimer);
        }

        kernel<<<h_gridDim, h_blockDim>>>(d_in, height, width, d_out);
    }

    // Get time
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    double time = sdkGetTimerValue(&hTimer) / (double)ITERATIONS;
    printf("Kernel time = %.5f ms\n", time);
    int nBlocks = gridXSize*gridYSize;
    int nThreads = nBlocks*block_size*block_size*nextpow2;
    printf("#Blocks=%d, #Threads=%d, Time/Thread=%f\n",
           nBlocks, nThreads, time*1000000.0/(double)nThreads);

    // Copy result back to host
    checkCudaErrors(hipMemcpy(h_out, d_out, img_size, hipMemcpyDeviceToHost));

    // write image to file for displaying
    save_image_bw("output.png", h_out, height, width);

    // Free device data
    hipFree(d_in);
    hipFree(d_out);

    // Free host data
    free(h_filter);
    free(h_out);
}
