#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#include "imgutils.h"

#define RADIUS 1
#define FILTER_SIZE ((RADIUS * 2) + 1)
#define BLOCK_SIZE 16
#define ITERATIONS 128

#define PRINT 1
#define RANDOM 1

// Constant memory for filter
// Since constant memory is read only and has its own cache, this improves the
// speed of accessing the filter
__constant__ float c_filter[FILTER_SIZE*FILTER_SIZE];

__global__ void kernel(float* d_in, int height, int width, float* d_out) {

    // Get global position in grid
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // actual location within image data
    // since image data is interleaved RGB values, offset like you would a 2D
    // image, multiply that by the number of channels (3) and add the z value
    // representing whether the pixel is R, G, or B
    unsigned int loc = (y * width) + x;

    // Shared memory block is big enough for the full block plus one radius of
    // pixels on all sides
    __shared__ float sh_data[BLOCK_SIZE + 2*RADIUS][BLOCK_SIZE + 2*RADIUS];

    // Copy to shared memory
    // It would be trivial for each thread to bring its corresponding pixel into
    // shared memory but the pixels in the radius around each block must also be
    // copied to shared memory.
    //
    // So, the following scheme is used to bring all pixels in.
    //
    // Each pixel copies four pixels, which are the four corners one RADIUS away
    // For example, with RADIUS=2, O is this thread's pixel and the X's are the
    // pixels brought into shared memory
    //      0  1  2  3  4
    //
    //0     X           X
    //1
    //2           O
    //3
    //4     X           X
    //
    // Note that RADIUS must be <= BLOCK_SIZE for the scheme to work.
    // If RADIUS > BLOCK_SIZE, each thread must pull in more than 4 pixels,
    // which requires a more expensive copying scheme.
    int x_tmp = x - RADIUS;
    int y_tmp = y - RADIUS;
    sh_data[threadIdx.x][threadIdx.y] = 
        (x_tmp < 0 || y_tmp < 0) ? 
        0 : 
        d_in[loc - RADIUS - width*RADIUS];

    x_tmp = x + RADIUS;
    sh_data[threadIdx.x + 2*RADIUS][threadIdx.y] = 
        (x_tmp >= width || y_tmp < 0) ?
        0 :
        d_in[loc + RADIUS - width*RADIUS];

    x_tmp = x - RADIUS;
    y_tmp = y + RADIUS;
    sh_data[threadIdx.x][threadIdx.y + 2*RADIUS] =
        (x_tmp < 0 || y_tmp >= height) ?
        0 :
        d_in[loc - RADIUS + width*RADIUS];

    x_tmp = x + RADIUS;
    sh_data[threadIdx.x + 2*RADIUS][threadIdx.y + 2*RADIUS] =
        (x_tmp >= width || y_tmp >= height) ?
        0 :
        d_in[loc + RADIUS + width*RADIUS];

    __syncthreads();

    // sum of all element-wise multiplications
    float sum = 0;

    // only perform convolution on pixels within radius
    // Global memory use and O(N^2) loop in kernel kill performance
    if (x >= RADIUS && y >= RADIUS && x < (width - RADIUS) && y < (height - RADIUS)) {
#pragma unroll
        for (int i = -RADIUS; i <= RADIUS; ++i) {
#pragma unroll
            for (int j = -RADIUS; j <= RADIUS; ++j) {

                // filter location based just on x and y
                int filt_x     = i + RADIUS;
                int filt_y     = j + RADIUS;
                int filter_loc = filt_y * FILTER_SIZE + filt_x;

                // add element-wise product to accumulator
                sum += sh_data[threadIdx.x+RADIUS+i][threadIdx.y+RADIUS+j] * c_filter[filter_loc];
            }
        }

        // add pixel value to output
        d_out[loc] = sum;
    }
}

int main(int argc, char** argv) {
    if (argc < 2) {
        printf("Usage: ./naive_conv <image>\n");
        return 0;
    }

    // read in image
    cv::Mat h_in     = read_image_bw(argv[1]);
    int     height   = h_in.rows;
    int     width    = h_in.cols;

    // Declare image and filter variables for host and device
    float *h_filter, *h_out, *d_in, *d_out;

    // size to allocate for image and filter variables
    unsigned int img_size         = width * height * sizeof(float);
    unsigned int full_filter_size = FILTER_SIZE * FILTER_SIZE * sizeof(float);

#if PRINT
    printf("img_size=%u, full_filter_size=%u\n", img_size, full_filter_size);
#endif

    // Allocate host data
    h_filter = (float*)malloc(full_filter_size);
    h_out    = (float*)malloc(img_size);

    // copy filter template to actual filter (maybe redundant)
#if RANDOM
    srand(200);
#else
    // Initialize filter template
    // clang-format off
    const float filt_template[FILTER_SIZE][FILTER_SIZE] = {
        {1,   1, 1},
        {1,  -8, 1},
        {1,   1, 1}
    };
    // clang-format on
#endif

    for (int row = 0; row < FILTER_SIZE; ++row) {
        for (int col = 0; col < FILTER_SIZE; ++col) {
            int idx = row * FILTER_SIZE + col;
#if RANDOM
            h_filter[idx] = (float)(rand() % 16);
#else
            h_filter[idx] = filt_template[row][col];
#endif
        }
    }

    // Allocate device data
    checkCudaErrors(hipMalloc((void**)&d_in, img_size));
    checkCudaErrors(hipMalloc((void**)&d_out, img_size));

    // Copy host memory to device
    checkCudaErrors(hipMemcpy(d_in, h_in.data, img_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_filter), h_filter, full_filter_size));

    // Let grid size be based on block size
    // Have just enough blocks to cover whole image
    // The -1 is to cover the case where image dimensions are multiples of
    // BLOCKS_SIZE
    int gridXSize = 1 + ((width - 1) / BLOCK_SIZE);
    int gridYSize = 1 + ((height - 1) / BLOCK_SIZE);
#if PRINT
    printf("gridXSize=%d, gridYSize=%d, BLOCK_SIZE=%d\n", 
           gridXSize, gridYSize, BLOCK_SIZE);
#endif
    dim3 h_gridDim(gridXSize, gridYSize);
    dim3 h_blockDim(BLOCK_SIZE, BLOCK_SIZE);

    // Run on GPU 0
    hipSetDevice(0);

    // Timing stuff
    StopWatchInterface *hTimer = NULL;
    sdkCreateTimer(&hTimer);

    // Kernel call
    // i=-1 is the warm up iteration
    for (int i = -1; i < ITERATIONS; ++i) {

        if (i == 0) {
            checkCudaErrors(hipDeviceSynchronize());
            sdkResetTimer(&hTimer);
            sdkStartTimer(&hTimer);
        }

        kernel<<<h_gridDim, h_blockDim>>>(d_in, height, width, d_out);
    }

    // Get time
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    double time = sdkGetTimerValue(&hTimer) / (double)ITERATIONS;
    printf("Kernel time = %.5f ms\n", time);
    int nBlocks  = gridXSize*gridYSize;
    int nThreads = nBlocks*BLOCK_SIZE*BLOCK_SIZE;
    printf("#Blocks=%d, #Threads=%d, Time/Thread=%f\n",
           nBlocks, nThreads, time*1000000.0/(double)nThreads);

    // Copy result back to host
    checkCudaErrors(hipMemcpy(h_out, d_out, img_size, hipMemcpyDeviceToHost));

    // write image to file
    save_image_bw("output.png", h_out, height, width);

    // Free device data
    hipFree(d_in);
    hipFree(d_out);

    // Free host data
    free(h_filter);
    free(h_out);
}
