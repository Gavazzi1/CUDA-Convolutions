#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#include "imgutils.h"

#define RADIUS 1
#define FILTER_SIZE ((RADIUS * 2) + 1)
#define BLOCK_SIZE 16
#define ITERATIONS 128

#define PRINT 1
#define RANDOM 0

__global__ void kernel(float* d_in, int height, int width, int channels, float* filter, float* d_out) {
    // shared memory block - accessible to all threads in a block
    // for each pixel in a block, you need access to all pixels in the block
    // plus one radius of pixels on all sides
    __shared__ float sh_data[BLOCK_SIZE + 2*RADIUS][BLOCK_SIZE + 2*RADIUS];

    // Get global position in grid
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = threadIdx.z;

    // actual location within image data
    // since image data is interleaved RGB values, offset like you would a 2D
    // image, multiply that by the number of channels (3) and add the z value
    // representing whether the pixel is R, G, or B
    unsigned int loc = channels * (y * width + x) + z;

    // Copy into shared memory
    sh_data[threadIdx.x][threadIdx.y] = (x-RADIUS < 0 || y-RADIUS < 0) ?
                                        0 :
                                        d_in[];

    // sum of all element-wise multiplications
    float sum = 0;

    // only perform convolution on pixels within radius
    // Global memory use and O(N^2) loop in kernel kill performance
    if (x >= RADIUS && y >= RADIUS && x < (width - RADIUS) && y < (height - RADIUS)) {
        int img_z = z;
#pragma unroll
        for (int i = -RADIUS; i <= RADIUS; ++i) {
#pragma unroll
            for (int j = -RADIUS; j <= RADIUS; ++j) {
                // x, y, and global location adjusted for filter radius
                int img_x   = x + i;
                int img_y   = y + j;
                int img_loc = channels * (img_y * width + img_x) + img_z;

                // filter location based just on x and y
                int filt_x     = i + RADIUS;
                int filt_y     = j + RADIUS;
                int filter_loc = filt_y * FILTER_SIZE + filt_x;

                // add element-wise product to accumulator
                sum += d_in[img_loc] * filter[filter_loc];
            }
        }

        // add pixel value to output
        d_out[loc] = sum;

#if 0
        if ((d_in[loc] - 0.0) > 0.001) {
            printf("x=%d, y=%d, z=%d, loc=%d, d_in=%f, d_out=%f\n", 
                   x, y, z, loc, d_in[loc], d_out[loc]);
        }
#endif
    }
}

int main(int argc, char** argv) {
    if (argc < 2) {
        printf("Usage: ./naive_conv <image>\n");
        return 0;
    }

    // read in image
    cv::Mat h_in     = read_image_bw(argv[1]);
    int     height   = h_in.rows;
    int     width    = h_in.cols;
    int     channels = h_in.channels();

#if PRINT
    printf("width=%d, height=%d, channels=%d, FILTER_SIZE=%d\n", 
           width, height, channels, FILTER_SIZE);
#endif

    // Declare image and filter variables for host and device
    float *h_filter, *h_out, *d_in, *d_filter, *d_out;

    // size to allocate for image and filter variables
    unsigned int img_size         = width * height * channels * sizeof(float);
    unsigned int full_filter_size = FILTER_SIZE * FILTER_SIZE * sizeof(float);

#if PRINT
    printf("img_size=%u, full_filter_size=%u\n", img_size, full_filter_size);
#endif

    // Allocate host data
    h_filter = (float*)malloc(full_filter_size);
    h_out    = (float*)malloc(img_size);

    // copy filter template to actual filter (maybe redundant)
#if RANDOM
    srand(200);
#else
    // Initialize filter template
    // clang-format off
    const float filt_template[FILTER_SIZE][FILTER_SIZE] = {
        {0, 0, 0},
        {0, 1, 0},
        {0, 0, 0}
    };
    // clang-format on
#endif

    for (int row = 0; row < FILTER_SIZE; ++row) {
        for (int col = 0; col < FILTER_SIZE; ++col) {
            int idx = row * FILTER_SIZE + col;
#if RANDOM
            h_filter[idx] = (float)(rand() % 16);
#else
            h_filter[idx] = filt_template[row][col];
#endif
        }
    }

    // Allocate device data
    checkCudaErrors(hipMalloc((void**)&d_in, img_size));
    checkCudaErrors(hipMalloc((void**)&d_filter, full_filter_size));
    checkCudaErrors(hipMalloc((void**)&d_out, img_size));

    // Copy host memory to device
    checkCudaErrors(hipMemcpy(d_in, h_in.data, img_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_filter, h_filter, full_filter_size, hipMemcpyHostToDevice));

    // Let grid size be based on block size
    // Have just enough blocks to cover whole image
    // The -1 is to cover the case where image dimensions are multiples of
    // BLOCKS_SIZE
    int gridXSize = 1 + ((width - 1) / BLOCK_SIZE);
    int gridYSize = 1 + ((height - 1) / BLOCK_SIZE);
#if PRINT
    printf("gridXSize=%d, gridYSize=%d, BLOCK_SIZE=%d\n", 
           gridXSize, gridYSize, BLOCK_SIZE);
#endif
    dim3 h_gridDim(gridXSize, gridYSize);
    dim3 h_blockDim(BLOCK_SIZE, BLOCK_SIZE, channels);

    // Run on GPU 0
    hipSetDevice(0);

    // Timing stuff
    StopWatchInterface *hTimer = NULL;
    sdkCreateTimer(&hTimer);

    // Kernel call
    for (int i = -1; i < ITERATIONS; ++i) {

        if (i == 0) {
            checkCudaErrors(hipDeviceSynchronize());
            sdkResetTimer(&hTimer);
            sdkStartTimer(&hTimer);
        }

        kernel<<<h_gridDim, h_blockDim>>>(d_in, height, width, channels, d_filter, d_out);
    }

    // Get time
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    double time = sdkGetTimerValue(&hTimer) / (double)ITERATIONS;
    printf("Kernel time = %.5f ms\n", time);

    // Copy result back to host
    checkCudaErrors(hipMemcpy(h_out, d_out, img_size, hipMemcpyDeviceToHost));

    // write image to file for displaying
    save_image_bw("output.png", h_out, height, width);

    // Free device data
    hipFree(d_in);
    hipFree(d_filter);
    hipFree(d_out);

    // Free host data
    free(h_filter);
    free(h_out);
}
