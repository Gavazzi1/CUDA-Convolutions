#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#include "imgutils.h"
#include "runkernel.h"

#define RADIUS 1
#define FILTER_SIZE ((RADIUS * 2) + 1)
#define BLOCK_SIZE 16
#define ITERATIONS 128

#define PRINT 1
#define RANDOM 0

int main(int argc, char** argv) {
    if (argc < 2) {
        printf("Usage: ./naive_conv <image>\n");
        return 0;
    }

    // read in image
    cv::Mat h_in     = read_image_bw(argv[1]);
    int     height   = h_in.rows;
    int     width    = h_in.cols;
    int     channels = h_in.channels();

#if PRINT
    printf("width=%d, height=%d, channels=%d, FILTER_SIZE=%d\n", 
           width, height, channels, FILTER_SIZE);
#endif

    // Declare image and filter variables for host and device
    float *h_filter, *h_out, *d_in, *d_filter, *d_out;

    // size to allocate for image and filter variables
    unsigned int img_size         = width * height * channels * sizeof(float);
    unsigned int full_filter_size = FILTER_SIZE * FILTER_SIZE * sizeof(float);

#if PRINT
    printf("img_size=%u, full_filter_size=%u\n", img_size, full_filter_size);
#endif

    // Allocate host data
    h_filter = (float*)malloc(full_filter_size);
    h_out    = (float*)malloc(img_size);

    // copy filter template to actual filter (maybe redundant)
#if RANDOM
    srand(200);
#else
    // Initialize filter template
    // clang-format off
    const float filt_template[FILTER_SIZE][FILTER_SIZE] = {
        {1, 1, 1},
        {1, -8, 1},
        {1, 1, 1}
    };
    // clang-format on
#endif

    for (int row = 0; row < FILTER_SIZE; ++row) {
        for (int col = 0; col < FILTER_SIZE; ++col) {
            int idx = row * FILTER_SIZE + col;
#if RANDOM
            h_filter[idx] = (float)(rand() % 16);
#else
            h_filter[idx] = filt_template[row][col];
#endif
        }
    }

    // Allocate device data
    checkCudaErrors(hipMalloc((void**)&d_in, img_size));
    checkCudaErrors(hipMalloc((void**)&d_filter, full_filter_size));
    checkCudaErrors(hipMalloc((void**)&d_out, img_size));

    // Copy host memory to device
    checkCudaErrors(hipMemcpy(d_in, h_in.data, img_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_filter, h_filter, full_filter_size, hipMemcpyHostToDevice));

    // Let grid size be based on block size
    // Have just enough blocks to cover whole image
    // The -1 is to cover the case where image dimensions are multiples of
    // BLOCKS_SIZE
    int gridXSize = 1 + ((width - 1) / BLOCK_SIZE);
    int gridYSize = 1 + ((height - 1) / BLOCK_SIZE);
#if PRINT
    printf("gridXSize=%d, gridYSize=%d, BLOCK_SIZE=%d\n", 
           gridXSize, gridYSize, BLOCK_SIZE);
#endif
    dim3 h_gridDim(gridXSize, gridYSize);
    dim3 h_blockDim(BLOCK_SIZE, BLOCK_SIZE, channels);

    // Run on GPU 0
    hipSetDevice(0);

    // Timing stuff
    StopWatchInterface *hTimer = NULL;
    sdkCreateTimer(&hTimer);

    // Kernel call
    for (int i = -1; i < ITERATIONS; ++i) {

        if (i == 0) {
            checkCudaErrors(hipDeviceSynchronize());
            sdkResetTimer(&hTimer);
            sdkStartTimer(&hTimer);
        }

        naivekernel(d_in, height, width, channels,
                  d_filter, RADIUS,
                  d_out,
                  h_gridDim, h_blockDim);
    }

    // Get time
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    double time = sdkGetTimerValue(&hTimer) / (double)ITERATIONS;
    printf("Kernel time = %.5f ms\n", time);

    // Copy result back to host
    checkCudaErrors(hipMemcpy(h_out, d_out, img_size, hipMemcpyDeviceToHost));

    // write image to file for displaying
    save_image_bw("output.png", h_out, height, width);

    // Free device data
    hipFree(d_in);
    hipFree(d_filter);
    hipFree(d_out);

    // Free host data
    free(h_filter);
    free(h_out);
}
